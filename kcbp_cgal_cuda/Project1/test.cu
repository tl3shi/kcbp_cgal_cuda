#include "hip/hip_runtime.h"
#include "common/book.h"
#include <vector>

#define MaxValue 100000000.0f
#define PI 3.1415926573

__host__ __device__ inline float dot(float3 &a, float3 &b)
{
    return a.x * b.x + a.y * b.y + a.z * b.z;
}


//each thread calculate one normal,max parallel degree is the normal_size
__global__ void projection(float* result , float3* normals, float3* points, int normal_size, int point_size)
{
    //int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idx = threadIdx.x;
    if(idx < normal_size)
    {
        //int result_index = -1;
        float3 normal = normals[idx];
        float distance = -MaxValue;
        for(int i = 0; i < point_size; i++)
        {
            float temp = dot(normal, points[i]);
            if(temp > distance)
            {
                distance = temp;
          //      result_index = i;
            }
        }
        result[idx] = distance;
    }
}

//each thread calcuate some points along normal
//use cpu to get the 
__global__ void projection1(float* result , float3* normals, float3* points, int normal_size, int point_size, const int BlockNum, const int ThreadNum)
{
    float3 normal;
    if(blockIdx.y < normal_size)
    {
        normal = normals[blockIdx.y];
        extern __shared__ float shared[];
        const int tid = threadIdx.x;
        const int bid = blockIdx.x;
        int i = 0;
        shared[tid] = -MaxValue;
        for(i = bid * ThreadNum + tid; i < point_size; i += BlockNum * ThreadNum)
        {
            float distance = dot(points[i],  normal);
            if (shared[tid] < distance)
            {
                shared[tid] = distance;//this should atom
            }
        }

        __syncthreads();
        if(tid == 0) //thread 0 to get local max distance in the block 
        {
            for(i = 1; i < ThreadNum; i++)
            {
                if(shared[0] < shared[i])
                    shared[0] = shared[i];
            }
            //result[normalid][bid]
            result[blockIdx.y * BlockNum + bid] = shared[0];//make_float2(blockIdx.y,bid);//shared[0];
            //result[bid * normal_size + blockIdx.y] = make_float2(bid, blockIdx.y);
        }
    }
}

__global__ void projection1Reduction(float* result , float3* normals, float3* points, int normal_size, int point_size, const int BlockNum, const int ThreadNum)
{
    float3 normal;
    if(blockIdx.y < normal_size)
    {
        normal = normals[blockIdx.y];
        extern __shared__ float shared[]; 
        const int tid = threadIdx.x;
        const int bid = blockIdx.x;
        shared[tid] = -MaxValue;
        for(int i = bid * ThreadNum + tid; i < point_size; i += BlockNum * ThreadNum)
        {
            float distance = dot(points[i],  normal);
            if (shared[tid]  < distance)
            {
                shared[tid] = distance;//this should atom
            }
        }

        __syncthreads();

        int nOffset;
        nOffset = ThreadNum /2;
        while(nOffset > 0)
        {
            if(tid < nOffset)
            {
                shared[tid] = shared[tid] > shared[tid + nOffset]  ? shared[tid] : shared[tid + nOffset];
            }
            nOffset >>= 1;
            __syncthreads();
        }
        if(tid == 0)
            result[blockIdx.y * BlockNum + bid] = shared[0];
    }
}

__global__ void projectionMulNormal(float* result , float3* normals, float3* points, int normal_size, int point_size, const int BlockNum, const int ThreadNum)
{
    extern __shared__ float shared[];
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
   
    //__shared__ float3 shared_normal;

    if(blockIdx.y < normal_size)
    {
        float3 shared_normal = normals[blockIdx.y];
        shared[tid] = -MaxValue;
    
        for(int i = bid * ThreadNum + tid; i < point_size; i += BlockNum * ThreadNum)
        {
            float distance = dot(points[i],  shared_normal);
            if (shared[tid] < distance)
            {
                shared[tid] = distance;//this should atom
            }
        }

        __syncthreads();
       
       #pragma region reduction
       if(BlockNum >= 512)
       {
            if(tid < 256)
            {
                shared[tid] = shared[tid] > shared[tid + 256]? shared[tid] : shared[tid + 256];
            }
             __syncthreads();
       }
   
       if(BlockNum >= 256)
       {
            if(tid < 128)
            {
                shared[tid] = shared[tid] > shared[tid + 128] ? shared[tid] : shared[tid + 128];
            }
             __syncthreads();
       }
 
       if(BlockNum >= 128)
       {
            if(tid < 64)
            {
                shared[tid] = shared[tid] > shared[tid + 64] ? shared[tid] : shared[tid + 64];
            }
             __syncthreads();
       }

       if(tid < 32)
       {
            if (BlockNum >=  64)
            {
                shared[tid] = shared[tid] > shared[tid + 32]  ? shared[tid] : shared[tid + 32];  
            }                                                
                                                             
            if (BlockNum >=  32)                             
            {                                                
                shared[tid] = shared[tid] > shared[tid + 16]  ? shared[tid] : shared[tid + 16];  
            }                                                

            if (BlockNum >=  16)
            {
                shared[tid] = shared[tid] > shared[tid + 8]  ? shared[tid] : shared[tid + 8];  
            }                                               
                                                            
            if (BlockNum >=   8)                            
            {                                               
                shared[tid] = shared[tid] > shared[tid + 4]  ? shared[tid] : shared[tid + 4];  
            }                                               
                                                            
            if (BlockNum >=   4)                            
            {                                               
                shared[tid] = shared[tid] > shared[tid + 2]  ? shared[tid] : shared[tid + 2];  
            }                                               
                                                            
            if (BlockNum >=   2)                            
            {                                               
                shared[tid] = shared[tid] > shared[tid + 1]  ? shared[tid] : shared[tid + 1];  
            }
       }
       #pragma endregion reduction

       if(tid == 0)
            result[blockIdx.y * BlockNum + bid] = shared[0];
    }
}

__global__ void projection1SingleNormal(float* result , float3 normal, float3* points, int point_size, const int BlockNum, const int ThreadNum)
{
    extern __shared__ float shared[];
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    int i = 0;
   
    __shared__ float3 shared_normal;
    shared_normal = normal;
    /* this is low throughput
    int nsize = point_size / ThreadNum;
    for(i = (bid * ThreadNum + tid)*nsize; i < (tid+1)*nsize; i++)
    {
        float distance = dot(points[i],  normal);
        if (shared[tid].x < distance)
        {
            shared[tid] = make_float2(distance, i);//this should atom
        }
    }*/
    
    shared[tid] = -MaxValue;
    for(i = bid * ThreadNum + tid; i < point_size; i += BlockNum * ThreadNum)
    {
        float distance = dot(points[i],  shared_normal);
        if (shared[tid] < distance)
        {
            shared[tid] = distance;//this should atom
        }
    }

    __syncthreads();

    
    /*
    if(tid == 0) //thread 0 to get local max distance in the block 
    {
        for(i = 1; i < ThreadNum; i++)
        {
            if(shared[0].x < shared[i].x)
                shared[0] = shared[i];
        }
        result[bid] = shared[0];
    }*/
    
    //this is better than above
    int nOffset;
    nOffset = ThreadNum /2;
    while(nOffset > 0)
    {
        if(tid < nOffset)
        {
            shared[tid] = shared[tid] > shared[tid + nOffset] ? shared[tid] : shared[tid + nOffset];
        }
        nOffset >>= 1;
        __syncthreads();
    }
    if(tid == 0)
        result[bid] = shared[0];
    
}

__global__ void projection1SingleNormal2(float* result , float3 normal, float3* points, int point_size, const int BlockNum, const int ThreadNum)
{
    extern __shared__ float shared[];
    const int tid = threadIdx.x;
    const int bid = blockIdx.x;
    //int i = 0;
   
    __shared__ float3 shared_normal;
    shared_normal = normal;
    shared[tid] = -MaxValue ;

   
    for(int i = bid * ThreadNum + tid; i < point_size; i += BlockNum * ThreadNum)
    {
        float distance = dot(points[i],  shared_normal);
        if (shared[tid]  < distance)
        {
            shared[tid] = distance;//this should atom
        }
    }

    __syncthreads();

   if(BlockNum >= 512)
   {
        if(tid < 256)
        {
            shared[tid] = shared[tid]  > shared[tid + 256]   ? shared[tid] : shared[tid + 256];
        }                                                   
         __syncthreads();                                   
   }                                                        
                                                            
   if(BlockNum >= 256)                                      
   {                                                        
        if(tid < 128)                                       
        {                                                   
            shared[tid] = shared[tid]  > shared[tid + 128]   ? shared[tid] : shared[tid + 128];
        }                                                   
         __syncthreads();                                  
   }                                                       
                                                           
   if(BlockNum >= 128)                                     
   {                                                       
        if(tid < 64)                                       
        {                                                  
            shared[tid] = shared[tid]  > shared[tid + 64] ? shared[tid] : shared[tid + 64];
        }                                                
         __syncthreads();                                
   }                                                     
                                                         
   if(tid < 32)                                          
   {                                                     
        if (BlockNum >=  64)                             
        {                                                
            shared[tid] = shared[tid]  > shared[tid + 32] ? shared[tid] : shared[tid + 32];  
        }                             
                                      
        if (BlockNum >=  32)          
        {                             
            shared[tid] = shared[tid]  > shared[tid + 16] ? shared[tid] : shared[tid + 16];  
        }

        if (BlockNum >=  16)
        {
            shared[tid] = shared[tid] > shared[tid + 8] ? shared[tid] : shared[tid + 8];  
        }                                               
                                                        
        if (BlockNum >=   8)                            
        {                                               
            shared[tid] = shared[tid] > shared[tid + 4] ? shared[tid] : shared[tid + 4];  
        }                                               
                                                        
        if (BlockNum >=   4)                            
        {                                               
            shared[tid] = shared[tid] > shared[tid + 2] ? shared[tid] : shared[tid + 2];  
        }                                               
                                                        
        if (BlockNum >=   2)                            
        {                                               
            shared[tid] = shared[tid] > shared[tid + 1] ? shared[tid] : shared[tid + 1];  
        }
   }

   if(tid == 0)
        result[bid] = shared[0];
}


void RunKernel(vector<float3>& normals, vector<float3>& points, vector<float> &result, int3 config)
{
    float3* d_normals;
    float3* d_points;
    float* d_result;

    HANDLE_ERROR(hipMalloc(reinterpret_cast<void**>(&d_points), points.size() * sizeof(float3)));
    HANDLE_ERROR(hipMalloc(reinterpret_cast<void**>(&d_normals), normals.size() * sizeof(float3)));
    HANDLE_ERROR(hipMalloc(reinterpret_cast<void**>(&d_result), normals.size() * sizeof(float)));
    
    // Copy data to the device
    hipMemcpy(d_points, &points[0], points.size() * sizeof(float3), hipMemcpyHostToDevice);
    hipMemcpy(d_normals, &normals[0], normals.size() * sizeof(float3), hipMemcpyHostToDevice);

    //int* result , float3* normals, float3* points, int normal_size, int point_size
    //gridsize, threadsize
    projection<<<config.x, config.y>>>(d_result, d_normals, d_points, normals.size(), points.size());

    // Copy back to the host
    hipMemcpy(&result[0], d_result, normals.size() * sizeof(float), hipMemcpyDeviceToHost);
    
    // Free device memory
    hipFree(d_points);
    hipFree(d_normals);
    hipFree(d_result);
}
extern "C"
void RunKernel1(vector<float3>& normals, vector<float3>& points, vector<float> &result, int blocksize,  int threadsize, float &elapsedTime, int &cpuclock)
{
    float3* d_normals;
    float3* d_points;
    float* d_result;

    unsigned int d_points_size = points.size() * sizeof(float3);
    unsigned int d_normals_size = normals.size() * sizeof(float3);
    
    unsigned int d_result_size = normals.size() * blocksize * sizeof(float);

    unsigned int d_shared_size = threadsize * sizeof(float);

    HANDLE_ERROR(hipMalloc(reinterpret_cast<void**>(&d_points),  d_points_size));
    HANDLE_ERROR(hipMalloc(reinterpret_cast<void**>(&d_normals), d_normals_size));
    HANDLE_ERROR(hipMalloc(reinterpret_cast<void**>(&d_result), d_result_size));

    // Copy data to the device
    hipMemcpy(d_points, &points[0], d_points_size, hipMemcpyHostToDevice);
    hipMemcpy(d_normals, &normals[0], d_normals_size, hipMemcpyHostToDevice);
    
    //gridsize, threadsize
  
    dim3 blockconfig(blocksize, normals.size(), 1);
    dim3 threadconfig(threadsize, 1, 1);
    
    hipEvent_t start_event, end_event; 
    HANDLE_ERROR(hipEventCreate(&start_event));
    HANDLE_ERROR(hipEventCreate(&end_event));
    HANDLE_ERROR(hipEventRecord(start_event, 0));

    //projection1<<<blockconfig, threadconfig, d_shared_size>>>(d_result, d_normals, d_points, normals.size(), points.size(), blocksize, threadsize);
     projection1Reduction<<<blockconfig, threadconfig, d_shared_size>>>(d_result, d_normals, d_points, normals.size(), points.size(), blocksize, threadsize);
    //projectionMulNormal<<<blockconfig, threadconfig, d_shared_size>>>(d_result, d_normals, d_points, normals.size(), points.size(), blocksize, threadsize);
   
    vector<float> cpu_result(normals.size() * blocksize);
    hipMemcpy(&cpu_result[0], d_result, d_result_size, hipMemcpyDeviceToHost);
    
    HANDLE_ERROR(hipEventRecord(end_event, 0));
    HANDLE_ERROR(hipEventSynchronize(end_event));
   
    HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start_event, end_event));

    clock_t start_time, end_time;
    start_time = clock();
    result.resize(normals.size());
    for(unsigned int normal_index  = 0; normal_index < normals.size(); normal_index++)
    {
        result[normal_index] = - MaxValue;
        for(int blockindex = 0; blockindex < blocksize; blockindex++)
        {
            float tmp = cpu_result[normal_index * blocksize + blockindex];
            if(result[normal_index] < tmp)
            {
                result[normal_index] = tmp;
            }
        }
    }
    end_time = clock();
    cpuclock = end_time - start_time;
    HANDLE_ERROR(hipEventDestroy(start_event));
    HANDLE_ERROR(hipEventDestroy(end_event));
    // Free device memory
    hipFree(d_points);
    hipFree(d_normals);
    hipFree(d_result);
}


void RunKernelSingleNormal(vector<float3>& normals, vector<float3>& points, vector<float> &result, const int &blocksize, const int &threadsize, float &elapsedTime)
{
    float3* d_points;
    float* d_result;

    unsigned int d_points_size = points.size() * sizeof(float3);
    unsigned int d_normals_size = normals.size() * sizeof(float3);
    
    unsigned int d_result_size = blocksize * sizeof(float);

    unsigned int d_shared_size = threadsize * sizeof(float);

    HANDLE_ERROR(hipMalloc(reinterpret_cast<void**>(&d_points),  d_points_size));
    HANDLE_ERROR(hipMalloc(reinterpret_cast<void**>(&d_result), d_result_size));

    HANDLE_ERROR(hipMemcpy(d_points, &points[0], d_points_size, hipMemcpyHostToDevice));

    
    //gridsize, threadsize
    dim3 blockconfig(blocksize, 1, 1);
    dim3 threadconfig(threadsize, 1, 1);

    hipEvent_t start_event, end_event; 
    HANDLE_ERROR(hipEventCreate(&start_event));
    HANDLE_ERROR(hipEventCreate(&end_event));
    HANDLE_ERROR(hipEventRecord(start_event, 0));

    vector<float> cpu_result(blocksize);
    for(unsigned int normal_index = 0; normal_index < normals.size(); normal_index++)
    {
      projection1SingleNormal<<<blockconfig, threadconfig, d_shared_size>>>(d_result, normals[normal_index], d_points, points.size(), blocksize, threadsize);
       // projection1SingleNormal2<<<blockconfig, threadconfig, d_shared_size>>>(d_result, normals[normal_index], d_points, points.size(), blocksize, threadsize);
        
        HANDLE_ERROR(hipMemcpy(&cpu_result[0], d_result, d_result_size, hipMemcpyDeviceToHost));
    
        result[normal_index] = - MaxValue;
        for(int blockindex = 0; blockindex < blocksize; blockindex++)
        {
            float tmp = cpu_result[blockindex];
            if(result[normal_index] < tmp)
            {
                result[normal_index] = tmp;
            }
        }
    }

    HANDLE_ERROR(hipEventRecord(end_event, 0));
    HANDLE_ERROR(hipEventSynchronize(end_event));
    HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start_event, end_event));

    HANDLE_ERROR(hipEventDestroy(start_event));
    HANDLE_ERROR(hipEventDestroy(end_event));

    // Free device memory
    hipFree(d_points);
    hipFree(d_result);
}
